#include "hip/hip_runtime.h"
#include "sort.cuh"
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>

__device__ void swap(int& a, int& b) {
    int t = a;
    a = b;
    b = t;
}

__global__ void bitonic_sort(int* arr, int i, int j) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int id_comp = id ^ j;
    if (id > id_comp)
        if ((arr[id] < arr[id_comp]) == !(id & i))
            swap(arr[id], arr[id_comp]);
}

void sort(float& total_time, std::vector <int> &nums) {
    int n = nums.size();
    int* input = NULL;
    assert(hipMalloc((void**)&input, n * sizeof(int)) == hipSuccess);
    assert(hipMemcpy(input, &nums[0], n * sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
    const int threadSize = 1024;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 2; i <= n; i *= 2)
        for (int j = i / 2; j; j /= 2)
            bitonic_sort<<<(n - 1) / threadSize + 1, threadSize>>>(input, i, j);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    assert(hipGetLastError() == hipSuccess);
    assert(hipMemcpy(&nums[0], input, n * sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipFree(input) == hipSuccess);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time(ms): %f\n", milliseconds);
    total_time += milliseconds;
}