#include "hip/hip_runtime.h"
#include "mul.cuh"
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>

__global__ void mul(const float* A, const float* B, std::size_t N, std::size_t M, std::size_t K, float* C) {
    __shared__ float tmp[256];
    float res[8][8];
    int x = threadIdx.x, y = threadIdx.y;
    int X = blockIdx.x * (blockDim.x * 8), Y = blockIdx.y * (blockDim.y * 8);
    int id = x * 16 + y;
    for (int i = 0; i < 8; ++i)
        for (int j = 0; j < 8; ++j) 
            res[i][j] = 0;
    for (int i = 0; i < M; ++i) {
        tmp[id] = (id < 128? A[(X + id) * M + i]: B[i * K + Y + id - 128]);
        __syncthreads();
        for (int j = 0; j < 8; ++j)
            for (int k = 0; k < 8; ++k)
                res[j][k] += tmp[x * 8 + j] * tmp[y * 8 + k + 128];
        __syncthreads();
    }
    for (int i = 0; i < 8; ++i)
        for (int j = 0; j < 8; ++j)
            C[(X + x * 8 + i) * K + Y + y * 8 + j] = res[i][j];
}

void matmul(float& total_time, const float* A, const float* B, std::size_t n, std::size_t m, std::size_t k, float* C) {
    float* a = NULL;
    assert(hipMalloc((void**)&a, n * m * sizeof(float)) == hipSuccess);
    float* b = NULL;
    assert(hipMalloc((void**)&b, m * k * sizeof(float)) == hipSuccess);
    assert(hipMemcpy(a, A, n * m * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(b, B, m * k * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    float* c = NULL;
    assert(hipMalloc((void**)&c, n * k * sizeof(float)) == hipSuccess);
    dim3 threadsPerBlock(16, 16);
    dim3 numblock((n - 1) / 128 + 1, (k - 1) / 128 + 1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    mul<<<numblock, threadsPerBlock>>>(a, b, n, m, k, c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    auto err = hipGetLastError();
    printf("Error: %s\n", hipGetErrorString(err));
    assert(err == hipSuccess);
    assert(hipMemcpy(C, c, n * k * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipFree(a) == hipSuccess);
    assert(hipFree(b) == hipSuccess);
    assert(hipFree(c) == hipSuccess);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time(ms): %f\n", milliseconds);
    total_time += milliseconds;
}