#include "hip/hip_runtime.h"
#include "add.cuh"
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>

__global__ void reduce_sum(const float* input_vecs, std::size_t n, std::size_t dim, float* output_vec) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < dim) {
        output_vec[i] = 0;
        for (int j = 0; j < n; ++j) 
            output_vec[i] += input_vecs[j * dim + i];
    }
}

void reduce(float& total_time, const float* input_vecs, std::size_t n, std::size_t dim, float* output_vec) {
    float* input = NULL;
    assert(hipMalloc((void**)&input, n * dim * sizeof(float)) == hipSuccess);
    float* output = NULL;
    assert(hipMalloc((void**)&output, dim * sizeof(float)) == hipSuccess);
    assert(hipMemcpy(input, input_vecs, n * dim * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    int threadsPerBlock = 1024;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_sum<<<(dim - 1) / threadsPerBlock + 1, threadsPerBlock>>>(input, n, dim, output);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    assert(hipGetLastError() == hipSuccess);
    assert(hipMemcpy(output_vec, output, dim * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipFree(input) == hipSuccess);
    assert(hipFree(output) == hipSuccess);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time(ms): %f\n", milliseconds);
    total_time += milliseconds;
}