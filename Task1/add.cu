#include "hip/hip_runtime.h"
#include "add.cuh"
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>

#define Float4(val) *(float4*)(&(val))

__global__ void reduce_sum(const float* input_vecs, std::size_t n, std::size_t dim, float* output_vec) {
    int i = 4 * (blockDim.x * blockIdx.x + threadIdx.x);
    if (i < dim) {
        float4 sum, a;
        sum.x = sum.y = sum.z = sum.w = 0;
        for (int j = 0; j < n; ++j) {
            a = Float4(input_vecs[j * dim + i]);
            sum.x += a.x;
            sum.y += a.y;
            sum.z += a.z;
            sum.w += a.w;
        }
        Float4(output_vec[i]) = sum;
    }
}

void reduce(float& total_time, const float* input_vecs, std::size_t n, std::size_t dim, float* output_vec) {
    float* input = NULL;
    assert(hipMalloc((void**)&input, n * dim * sizeof(float)) == hipSuccess);
    float* output = NULL;
    assert(hipMalloc((void**)&output, dim * sizeof(float)) == hipSuccess);
    assert(hipMemcpy(input, input_vecs, n * dim * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    int threadsPerBlock = 32;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_sum<<<(dim - 1) / (threadsPerBlock / 4) + 1, threadsPerBlock>>>(input, n, dim, output);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    assert(hipGetLastError() == hipSuccess);
    assert(hipMemcpy(output_vec, output, dim * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipFree(input) == hipSuccess);
    assert(hipFree(output) == hipSuccess);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU time(ms): %f\n", milliseconds);
    total_time += milliseconds;
}