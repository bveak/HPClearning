#include "hip/hip_runtime.h"
#include "add.cuh"
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>

__global__ void reduce_sum(const float* input_vecs, std::size_t n, std::size_t dim, float* output_vec) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < dim) {
        output_vec[i] = 0;
        for (int j = 0; j < n; ++j) 
            output_vec[i] += input_vecs[j * dim + i];
    }
}

void reduce(const float* input_vecs, std::size_t n, std::size_t dim, float* output_vec) {
    float* input = NULL;
    assert(hipMalloc((void**)&input, n * dim * sizeof(float)) == hipSuccess);
    float* output = NULL;
    assert(hipMalloc((void**)&output, dim * sizeof(float)) == hipSuccess);
    assert(hipMemcpy(input, input_vecs, n * dim * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    int threadsPerBlock = 1024;
    reduce_sum<<<(dim - 1) / threadsPerBlock + 1, threadsPerBlock>>>(input, n, dim, output);
    assert(hipGetLastError() == hipSuccess);
    auto err = hipMemcpy(output_vec, output, dim * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        assert(err != hipSuccess);
        fprintf(stderr, "Failed to reduce sum (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // assert(hipMemcpy(output_vec, output, dim, hipMemcpyDeviceToHost) == hipSuccess);
    assert(hipFree(input) == hipSuccess);
    assert(hipFree(output) == hipSuccess);
}