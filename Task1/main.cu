#include "hip/hip_runtime.h"
#include "add.cuh"
#include <cstdio>
#include <cassert>
#include <random>
#include <cmath>

__global__ void warmup_kernel(int* test) {
    int i = 1, j = 1;
    i += j;
    if (blockIdx.x == threadIdx.x)
        ++test[blockIdx.x];
}

void warmup() {
    int* test = NULL;
    assert(hipMalloc((void**)&test, 1024 * sizeof(int)) == hipSuccess);
    int* rnd = (int*)malloc(1024 * sizeof(int));
    unsigned rndval = 1;
    for (int i = 0; i < 1024; ++i) {
        rnd[i] = rndval;
        rndval ^= rndval << 3;
        rndval ^= rndval >> 5;
        rndval ^= rndval << 17;
    }
    assert(hipMemcpy(test, rnd, 1024 * sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
    for (int i = 0; i < 8; ++i) {
        warmup_kernel<<<1024, 1024>>>(test);
        assert(hipGetLastError() == hipSuccess);
    }
    assert(hipMemcpy(rnd, test, 1024 * sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);
}

int main() {
    std::size_t n = 50000 << 9;
    std::size_t dim = 32;
    std::mt19937 rnd((0x3ac2ed7b));
    float* input = (float*)malloc(n * dim * sizeof(float));
    for (int i = 0; i < n * dim; ++i)
        input[i] = rnd() / 1e4;
    float* output = (float*)malloc(dim * sizeof(float));
    float total_time = 0, max_time = 0, min_time = 1e5;
    warmup();
    int TestCount = 10;
    for (int i = 0; i < TestCount; ++i) {
        float current_time = 0;
        reduce(current_time, input, n, dim, output);
        max_time = std::max(max_time, current_time);
        min_time = std::min(min_time, current_time);
        total_time += current_time;
    }
    printf("GPU average time(ms): %f\n", (total_time - max_time - min_time) / (TestCount - 2));
    float* ansput = (float*)malloc(dim * sizeof(float));
    for (int i = 0; i < dim; ++i)
        ansput[i] = 0;
    time_t start = clock();
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < dim; ++j)
            ansput[j] += input[i * dim + j];
    }
    time_t end = clock();
    printf("CPU time(ms): %f\n", 1.0 * (end - start) / CLOCKS_PER_SEC * 1e3);
    // printf("%f %f\n", ansput[0], output[0]);
    for (int i = 0; i < dim; ++i)
        assert(std::fabs((ansput[i] - output[i]) / ansput[i]) < 1e-3);
    puts("Accepted!!");
    free(input);
    free(output);
    return 0;
}