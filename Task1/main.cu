#include "hip/hip_runtime.h"
#include "add.cuh"
#include <cstdio>
#include <cassert>
#include <random>
#include <cmath>

__global__ void warmup_kernel(int* test) {
    int i = 1, j = 1;
    i += j;
    if (blockIdx.x == threadIdx.x)
        ++test[blockIdx.x];
}

void warmup() {
    int* test = NULL;
    assert(hipMalloc((void**)&test, 1024 * sizeof(int)) == hipSuccess);
    int* rnd = (int*)malloc(1024 * sizeof(int));
    unsigned rndval = 1;
    for (int i = 0; i < 1024; ++i) {
        rnd[i] = rndval;
        rndval ^= rndval << 3;
        rndval ^= rndval >> 5;
        rndval ^= rndval << 17;
    }
    assert(hipMemcpy(test, rnd, 1024 * sizeof(int), hipMemcpyHostToDevice) == hipSuccess);
    for (int i = 0; i < 8; ++i)
        warmup_kernel<<<1024, 1024>>>(test);
    assert(hipMemcpy(rnd, test, 1024 * sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);
}

int main() {
    std::size_t n = 50000;
    std::size_t dim = 16384;
    std::mt19937 rnd((0x3ac2ed7b));
    float* input = (float*)malloc(n * dim * sizeof(float));
    for (int i = 0; i < n * dim; ++i)
        input[i] = rnd() / 1e4;
    float* output = (float*)malloc(dim * sizeof(float));
    float total_time = 0;
    warmup();
    int TestCount = 10;
    for (int i = 0; i < TestCount; ++i) {
        reduce(total_time, input, n, dim, output);
    }
    printf("GPU average time(ms): %f\n", total_time / TestCount);
    float* ansput = (float*)malloc(dim * sizeof(float));
    for (int i = 0; i < dim; ++i)
        ansput[i] = 0;
    time_t start = clock();
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < dim; ++j)
            ansput[j] += input[i * dim + j];
    }
    time_t end = clock();
    printf("CPU time(ms): %f\n", 1.0 * (end - start) / CLOCKS_PER_SEC * 1e3);
    for (int i = 0; i < dim; ++i)
        assert(std::fabs(ansput[i] - output[i]) < 1e-5);
    puts("Accepted!!");
    free(input);
    free(output);
    return 0;
}